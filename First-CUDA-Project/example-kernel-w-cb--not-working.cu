#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <algorithm>
#include <chrono>
#include <iostream>

//#include <json.hpp>
#include <cxxopts.hpp>
#include <rapidjson/rapidjson.h>
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <rapidjson/document.h>
#include <rapidjson/istreamwrapper.h>
#include <rapidjson/writer.h>
#include <rapidjson/stringbuffer.h>
#include <rapidjson/ostreamwrapper.h>
#include <algorithm> 
#include <list>
#include "nvToolsExt.h"
#include <limits> // press enter...
#include <Windows.h>


constexpr char KEY_ASYNC[] = "async";
constexpr char KEY_CHANNEL[] = "channel";
constexpr char KEY_CONFIG[] = "config";
constexpr char KEY_DURATION[] = "duration";
constexpr char KEY_DURATION_TYPE[] = "duration-type";
constexpr char KEY_HEIGHT[] = "height";
constexpr char KEY_HELP[] = "help";
constexpr char KEY_STREAMS[] = "streams";
constexpr char KEY_OUTPUT[] = "output";
constexpr char KEY_TRANSFER[] = "transfer";
constexpr char KEY_TYPE[] = "type";
constexpr char KEY_WIDTH[] = "width";

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
int checkCuda(hipError_t result)
{
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        return 1;
    }

    return 0;
}

class cLimitTimer
{
    std::chrono::time_point<std::chrono::steady_clock> Start;
    double Limit;
public:
    cLimitTimer(double limit) : Limit(limit)
    {
        Start = std::chrono::steady_clock::now();
    }
    ~cLimitTimer()
    {
        auto end = std::chrono::steady_clock::now();

        std::chrono::duration<double> diff = end - Start;
        if (diff.count() > Limit)
            std::cout << diff.count() << "s greater than limit " << Limit << "s\n";
        else if(diff.count() > 1.0)
            std::cout << diff.count() << "s\n";
        else
            std::cout << diff.count() * 1000 << "ms\n";

    }
};

template<class T>
__global__ void addKernel(T *dev, int size)
{
    for (int i = 0; i < size; i++)
        dev[i] += 1;
}

namespace application {
enum class Type {
    Invalid,
    Floating,
    Integer
};

Type stringToType(const std::string& key)
{
    Type result = Type::Invalid;

    std::map<std::string, Type> mapOfType = {
        {"floating", Type::Floating},
        {"integer", Type::Integer}
    };

    std::map<std::string, Type>::iterator it = mapOfType.find(key);

    if (it != mapOfType.end())
        result = it->second;
    
    return result;
}


enum class Transfer {
    Invalid,
    Pageable,
    Pinned
};

Transfer stringToTransfer(const std::string& key)
{
    Transfer result = Transfer::Invalid;

    std::map<std::string, Transfer> mapOfType = {
        {"pageable", Transfer::Pageable},
        {"pinned", Transfer::Pinned}
    };

    std::map<std::string, Transfer>::iterator it = mapOfType.find(key);

    if (it != mapOfType.end())
        result = it->second;

    return result;
}

enum class DurationType {
    Invalid,
    Counter,
    Timer
};

DurationType stringToDurationType(const std::string& key)
{
    DurationType result = DurationType::Invalid;

    std::map<std::string, DurationType> mapOfDurationType = {
        {"counter", DurationType::Counter},
        {"timer", DurationType::Timer}
    };

    std::map<std::string, DurationType>::iterator it = mapOfDurationType.find(key);

    if (it != mapOfDurationType.end())
        result = it->second;

    return result;
}

struct Parameters {
    int channel = -1;
    int duration = -1;
    DurationType durationType = DurationType::Invalid;
    int height = -1;
    std::string output{};
    int streams = -1;
    Transfer transfer = Transfer::Invalid;
    Type type = Type::Invalid;
    int width = -1;
    bool isAsync{ false };
};

struct Timers {
    float timerH2D = -1.0f;
    float timerD2H = -1.0f;
};
}

int readJsonFile(const std::string configFile, application::Parameters &parameters)
{
    using namespace rapidjson;

    std::ifstream ifs{ configFile };
    if (!ifs.is_open())
    {
        std::cerr << "Could not open " << configFile << " file!" << std::endl;
        return 1;
    }

    IStreamWrapper isw{ ifs };

    Document doc{};
    doc.ParseStream(isw);

    if (doc.HasParseError())
    {
        std::cerr << "Error  : " << doc.GetParseError() << '\n'
            << "Offset : " << doc.GetErrorOffset() << std::endl;
        return 1;
    }

    if(doc.HasMember(KEY_ASYNC) && doc[KEY_ASYNC].IsBool())
        parameters.isAsync = doc[KEY_ASYNC].GetBool();

    if (doc.HasMember(KEY_CHANNEL) && doc[KEY_CHANNEL].IsInt())
        parameters.channel = doc[KEY_CHANNEL].GetInt();

    if (doc.HasMember(KEY_DURATION) && doc[KEY_DURATION].IsInt())
        parameters.duration = doc[KEY_DURATION].GetInt();

    if (doc.HasMember(KEY_DURATION_TYPE) && doc[KEY_DURATION_TYPE].IsString())
    {
        auto durationType = doc[KEY_DURATION_TYPE].GetString();
        parameters.durationType = application::stringToDurationType(durationType);
    }

    if (doc.HasMember(KEY_HEIGHT) && doc[KEY_HEIGHT].IsInt())
        parameters.height = doc[KEY_HEIGHT].GetInt();

    if (doc.HasMember(KEY_STREAMS) && doc[KEY_STREAMS].IsInt())
        parameters.streams = doc[KEY_STREAMS].GetInt();

    if (doc.HasMember(KEY_TRANSFER) && doc[KEY_TRANSFER].IsString())
    {
        auto type = doc[KEY_TRANSFER].GetString();
        parameters.transfer = application::stringToTransfer(type);
    }

    if (doc.HasMember(KEY_TYPE) && doc[KEY_TYPE].IsString())
    {
        auto type = doc[KEY_TYPE].GetString();
        parameters.type = application::stringToType(type);
    }

    if (doc.HasMember(KEY_OUTPUT) && doc[KEY_OUTPUT].IsString())
    {
        auto output = doc[KEY_OUTPUT].GetString();
        parameters.output = doc[KEY_OUTPUT].GetString();
    }

    if (doc.HasMember(KEY_WIDTH) && doc[KEY_WIDTH].IsInt())
        parameters.width = doc[KEY_WIDTH].GetInt();

    return 0;
}

int validate(application::Parameters& parameters)
{
    int err = 0;

    if (parameters.channel < 1)
    {
        std::cerr << "Invalid channel option" << std::endl;
        err = 1;
    }
    else if (parameters.duration < 1)
    {
        std::cerr << "Invalid duration option" << std::endl;
        err = 1;
    } 
    else if (parameters.durationType == application::DurationType::Invalid)
    {
        std::cerr << "Invalid duration type option" << std::endl;
        err = 1;
    }
    else if (parameters.height < 1)
    {
        std::cerr << "Invalid height option" << std::endl;
        err = 1;
    }
    else if (parameters.streams < 1)
    {
        std::cerr << "Invalid streams option" << std::endl;
        err = 1;
    }
    else if (parameters.type == application::Type::Invalid)
    {
        std::cerr << "Invalid type option" << std::endl;
        err = 1;
    }
    else if (parameters.transfer == application::Transfer::Invalid)
    {
        std::cerr << "Invalid transfer option" << std::endl;
        err = 1;
    }
    else if (parameters.width< 1)
    {
        std::cerr << "Invalid width option" << std::endl;
        err = 1;
    }

    return err;
}

int applyOptions(cxxopts::ParseResult &result, application::Parameters& parameters)
{
    int err = 0;
    if (result.count(KEY_CONFIG))
    {
        std::string configFile = { result[KEY_CONFIG].as<std::string>() };
        err = readJsonFile(configFile, parameters);
    }

    if (err == 0)
    {
        if (result.count(KEY_ASYNC))
            parameters.isAsync = result[KEY_ASYNC].as<bool>();

        if (result.count(KEY_CHANNEL))
            parameters.channel = result[KEY_CHANNEL].as<int>();

        if (result.count(KEY_DURATION))
            parameters.duration = result[KEY_DURATION].as<int>();

        if (result.count(KEY_DURATION_TYPE))
        {
            auto durationType = result[KEY_DURATION_TYPE].as<std::string>();
            parameters.durationType = application::stringToDurationType(durationType);
        }

        if (result.count(KEY_HEIGHT))
            parameters.height = result[KEY_HEIGHT].as<int>();

        if (result.count(KEY_OUTPUT))
        {
            auto output = result[KEY_OUTPUT].as<std::string>();
            parameters.output = output;
        }

        if (result.count(KEY_STREAMS))
        {
            auto streams = result[KEY_STREAMS].as<int>();
            parameters.streams = streams;
        }

        if (result.count(KEY_TRANSFER))
        {
            auto transfer = result[KEY_TRANSFER].as<std::string>();
            parameters.transfer = application::stringToTransfer(transfer);
        }

        if (result.count(KEY_TYPE))
        {
            auto type = result[KEY_TYPE].as<std::string>();
            parameters.type = application::stringToType(type);
        }

        if (result.count(KEY_WIDTH))
            parameters.width = result[KEY_WIDTH].as<int>();
    }

    return err;
}

template <class T>
int copy(const application::Parameters& parameters, T* host, T* D2H, application::Timers &timers, hipStream_t& stream);

template <class T>
int startCounterLoop(const application::Parameters& parameters, T* host, T* D2H, std::list<application::Timers> &listOfTimers)
{
    const int duration = parameters.duration;
    const int streams = parameters.streams;
    int err = 0;

    std::list<hipStream_t*> streamList{};


    for (int i = 0; i < streams; i++)
    {
        auto stream = new hipStream_t();
        hipStreamCreate(stream);
        std::cout << stream << std::endl;
        streamList.push_back(stream);
    }
    
    std::cout << "size of streamList: " << streamList.size() << std::endl;
    for (auto* stream : streamList)
    {
        std::cout << stream << std::endl;
    }


    auto it = streamList.begin();
    for (int i = 0; i < duration; i++)
    {
        std::cout << i << ": " << (*it) << std::endl;
        application::Timers timers;

        if (err = copy(parameters, host, D2H, timers, *(*it)), err)
            break;

        listOfTimers.push_back(timers);

        if (++it == streamList.end())
            it = streamList.begin();
    }

    for (auto* stream : streamList)
    {
        delete stream;
    }

    return err;
}

template <class T>
int startTimerLoop(application::Parameters& parameters, T* host, T* D2H, std::list<application::Timers> & listOfTimers)
{
    const int duration = parameters.duration;
    const int streams = parameters.streams;
    int err = 0;

    std::list<hipStream_t> streamList{};

    for (int i = 0; i < streams; i++)
    {
        streamList.push_back({});
        hipStream_t stream = *streamList.end();
        hipStreamCreate(&stream);
    };

    auto start = std::chrono::system_clock::now();
    auto end = std::chrono::system_clock::now();
    auto it = streamList.begin();
    while ((std::chrono::duration_cast<std::chrono::seconds>(end - start).count() != duration))
    {
        application::Timers timers;

        if (err = copy(parameters, host, D2H, timers, *it), err)
            break;

        listOfTimers.push_back(timers);

        if (++it == streamList.end())
            it = streamList.begin();

        end = std::chrono::system_clock::now();
    }

    return err;
}

void printToScreen(std::list<application::Timers> &listOfTimers)
{
    for (auto const& i : listOfTimers)
        std::cout << i.timerH2D << ", " << i.timerD2H << std::endl;
}

void printToFile(const std::string &output, std::list<application::Timers>& listOfTimers)
{
    std::ofstream outputFile(output);

    if (outputFile.is_open())
        for (auto const& i : listOfTimers)
            outputFile << i.timerH2D << ", " << i.timerD2H << std::endl;
    else
        std::cerr << "cannot open file!" << std::endl;
}

void print(application::Parameters& parameters, std::list<application::Timers>& listOfTimers)
{
    const auto output = parameters.output;

    if (output.size())
        printToFile(output, listOfTimers);
    else
        printToScreen(listOfTimers);
}

template <class T>
void fillArrays(application::Parameters& parameters, T* host, T* D2H)
{
    const int height = parameters.height;
    const int width = parameters.width;
    const int channels = parameters.channel;
    const int size = height * width * channels;

    for (int i = 0; i < size; ++i)
    {
        host[i] = static_cast<T>(1.0);
        D2H[i] = static_cast <T>(0.0);
    }
}

template <class T>
int startCopyTest(application::Parameters &parameters, std::list<application::Timers> &listOfTimers)
{
    const int height = parameters.height;
    const int width = parameters.width;
    const int channels = parameters.channel; 
    const int size = height * width * channels;
    int err = 0;

    T* host = nullptr;
    T* D2H = nullptr;
    
    // arrays allocation
    switch (parameters.transfer)
    {
        case application::Transfer::Pageable:
        {
            host = (T*)malloc(sizeof(T) * size);
            D2H = (T*)malloc(sizeof(T) * size);
            break;
        }
        case application::Transfer::Pinned:
        {
            if (err = checkCuda(hipHostMalloc((T**)&host, size * sizeof(T))), err)
            {
                break;
            }
            else if (err = checkCuda(hipHostMalloc((T**)&D2H, size * sizeof(T))), err)
            {
                break;
            }
            break;
        }
        case::application::Transfer::Invalid:
            err = 1;
    }
            

    if (host == nullptr)
    {
        std::cerr << "host allocation failed!" << std::endl;
        err = 1;
    }
    else if (D2H == nullptr)
    {
        std::cerr << "D2H allocation failed!" << std::endl;
        err = 1;
    }

    if (!err)
    {
        fillArrays(parameters, host, D2H);

        if (parameters.durationType == application::DurationType::Counter && !err)
            err = startCounterLoop<T>(parameters, host, D2H, listOfTimers);
        else if (parameters.durationType == application::DurationType::Timer && !err)
            err = startTimerLoop<T>(parameters, host, D2H, listOfTimers);

        // free allocations
        switch (parameters.transfer)
        {
        case application::Transfer::Pageable:
        {
            if (host)
                free(host);
            if (D2H)
                free(D2H);
            break;
        }
        case application::Transfer::Pinned:
        {
            if (host)
                hipHostFree(host);
            if (D2H)
                hipHostFree(D2H);
            break;
        }
        case::application::Transfer::Invalid:
            err = 1;
        }
    }
    
    
    return err;
}

int process(application::Parameters& parameters, std::list<application::Timers>& listOfTimers)
{
    int err = 0;
    auto type = parameters.type;

    switch (type)
    {
        case application::Type::Floating:
            err = startCopyTest<float>(parameters, listOfTimers);
            break;
        case application::Type::Integer:
            err = startCopyTest<int>(parameters, listOfTimers);
            break;
        case application::Type::Invalid:
            std::cerr << "Invalid type!" << std::endl;
            err = 1;
            break;
    }

    return err;
}


void print(application::Parameters &parameters)
{
    // enum to string are limited.
    std::cout << "isAsync: " << (parameters.isAsync ? "true" : "false") << std::endl;
    std::cout << "channel: " << parameters.channel << std::endl;
    std::cout << "duration: " << parameters.duration << std::endl;
    std::cout << "duration-type: " << (parameters.durationType == application::DurationType::Timer ? "Timer" : "Counter") << std::endl;
    std::cout << "height: " << parameters.height << std::endl;
    std::cout << "output: " << parameters.output << std::endl;
    std::cout << "streams: " << parameters.streams << std::endl;
    std::cout << "transfer: " << (parameters.transfer  == application::Transfer::Pageable ? "Pageable" : "Pinned")<< std::endl;
    std::cout << "type: " << (parameters.type == application::Type::Floating ? "Floating" : "Integer") << std::endl;
    std::cout << "width: " << parameters.width << std::endl;
}

int main(int argc, char** argv)
{
    application::Parameters parameters;
    std::list<application::Timers> listOfTimers;
    int err = 0;

    cxxopts::ParseResult result;
    cxxopts::Options options("first-CUDA-Project", "copy back and forth from host to GPU an array and measure times.");

    options.add_options()
        (KEY_ASYNC, "Copy using streams (2).", cxxopts::value<bool>()->default_value("false"))
        (KEY_CHANNEL, "Number of channels for the array.", cxxopts::value<int>())
        (KEY_CONFIG, "Config file to load.", cxxopts::value<std::string>())
        (KEY_DURATION, "Number of iterations or time in seconds.", cxxopts::value<int>())
        (KEY_DURATION_TYPE, "Possible value is counter or timer.", cxxopts::value<std::string>())
        (KEY_HEIGHT, "Height of the array.", cxxopts::value<int>())
        (KEY_OUTPUT, "Store to file instead of print to screen.", cxxopts::value<std::string>())
        (KEY_STREAMS, "Number of streams.", cxxopts::value<int>()->default_value("1"))
        (KEY_TRANSFER, "Transfer memory mode. possible options are \"pageable\" and \"pinned\".", cxxopts::value<std::string>()->default_value("pageable"))
        (KEY_TYPE, "Possible value is floating or integer.", cxxopts::value<std::string>())
        (KEY_WIDTH, "Width of the array", cxxopts::value<int>())
        (KEY_HELP, "Print usage.");

    try
    {
        result = options.parse(argc, argv);
    }
    catch (...)
    {
        std::cerr << "Exception thrown from parse!" << std::endl;
    }

    if (result.count("help"))
        std::cout << options.help() << std::endl;
    else if (err = applyOptions(result, parameters), err)
        std::cerr << "(EE) applyOptions -> error" << std::endl;
    else if (err = validate(parameters), err)
        std::cerr << "(EE) validate -> error" << std::endl;
    else if (err = process(parameters, listOfTimers), err)
        std::cerr << "(EE) process -> error" << std::endl;
    else
    {
        print(parameters, listOfTimers);
        //print(parameters);
    }
    
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.

    err = checkCuda(hipDeviceReset());
    /*
    auto cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        std::cerr <<  "hipDeviceReset failed!" << std::endl;
        err = 1;
    }
    */

    std::cout << "Press Enter to ContinueX";
#undef max // for visual studio intellisense...
    std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');

   return err;
}

template <class T>
int copyCUDA(const application::Parameters& parameters, T* dst, T* src, enum hipMemcpyKind cpyKind, hipStream_t& stream)
{
    const int height = parameters.height;
    const int width = parameters.width;
    const int channels = parameters.channel;
    const int size = height * width * channels;

    int err = 0;

    if (parameters.isAsync)
    {

        err = checkCuda(hipMemcpyAsync(dst, src, size * sizeof(T), cpyKind, stream));
    }
    else
    {
        err = checkCuda(hipMemcpy(dst, src, size * sizeof(T), cpyKind));
    }

    return err;
}

struct UserData
{
    application::Timers* timers = nullptr;
    hipEvent_t* startH2D = nullptr;
    hipEvent_t* startD2H = nullptr;

    hipEvent_t* stopH2D = nullptr;
    hipEvent_t* stopD2H = nullptr;
};
void CUDART_CB callback(hipStream_t stream, hipError_t status, void* userData)
{
    std::cout << "TOTO" << std::endl;
    
    application::Timers* timers{ ((UserData*)userData)->timers };
    if (timers)
    {
        std::cout << timers << std::endl;
        hipEvent_t* stopH2D{ ((UserData*)userData)->stopH2D };


        if (stopH2D)
            std::cout << "J1: " << stopH2D << std::endl;

        //hipEventSynchronize(*stopH2D); NO


        hipEvent_t *startH2D{ ((UserData*)userData)->startH2D };

        float titi;
        hipEventElapsedTime(&titi, *startH2D, *stopH2D); // in ms

        std::cout << "H1: " << titi << std::endl;
    }

    /*hipEvent_t *startH2D{ ((UserData*)userData)->startH2D };
    hipEvent_t *stopH2D{ ((UserData*)userData)->stopH2D };
    hipEvent_t *startD2H{ ((UserData*)userData)->startD2H };
    hipEvent_t *stopD2H{ ((UserData*)userData)->stopD2H };


    hipEventElapsedTime(&timers->timerH2D, *startH2D, *stopH2D); // in ms
    hipEventElapsedTime(&timers->timerD2H, *startD2H,* stopD2H); // in ms
    */
}

template <class T>
int copy(const application::Parameters& parameters, T* host, T* D2H, application::Timers& timers, hipStream_t& stream)
{

    const int height = parameters.height;
    const int width = parameters.width;
    const int channels = parameters.channel;
    const int size = height * width * channels;
    const uint32_t COLOR_GREEN = 0xFF00FF00;

    //hipError_t cudaStatus;
    hipEvent_t startH2D, stopH2D;
    hipEvent_t startD2H, stopD2H;
    UserData userData{ &timers, &startH2D, &startD2H, &stopH2D, &stopD2H };
    hipEventCreate(userData.startH2D);
    hipEventCreate(userData.stopH2D);
    hipEventCreate(userData.startD2H);
    hipEventCreate(userData.stopD2H);

    T* dev = 0;
    int err = 0;
    nvtxEventAttributes_t eventAttrib = { 0 }; // zero the structure

    // Choose which GPU to run on, change this on a multi-GPU system.
    if (checkCuda(hipSetDevice(0)))
        err = 1;
    else if (checkCuda(hipMalloc((void**)&dev, size * sizeof(T))))
        err = 1;
    else
    {
        // Copy input vectors from host memory to GPU buffers.
        eventAttrib = { 0 }; // zero the structure
        eventAttrib.version = NVTX_VERSION;
        eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
        eventAttrib.colorType = NVTX_COLOR_ARGB;
        eventAttrib.color = COLOR_GREEN;
        eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII;
        eventAttrib.message.ascii = __FUNCTION__ ":timerH2D ";

        nvtxRangePushEx(&eventAttrib);
        hipEventRecord(*(userData.startH2D), stream);

        //err = checkCuda(hipMemcpy(dev, host, size * sizeof(T), hipMemcpyHostToDevice))
        err = copyCUDA<T>(parameters, dev, host, hipMemcpyHostToDevice, stream);

        hipEventRecord(*(userData.stopH2D), stream);
        nvtxRangePop();

    }

    // <Launch a kernel on the GPU here>
    addKernel<T> << <1, 1, 0, stream >> > (dev, size); // using thread, keep in mind to don't go over max cores.

    // Copy output vector from GPU buffer to host memory.
    if (!err)
    {
        eventAttrib = { 0 }; // zero the structure
        eventAttrib.version = NVTX_VERSION;
        eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
        eventAttrib.colorType = NVTX_COLOR_ARGB;
        eventAttrib.color = COLOR_GREEN;
        eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII;
        eventAttrib.message.ascii = __FUNCTION__ ":timerD2H ";

        nvtxRangePushEx(&eventAttrib);
        hipEventRecord(*(userData.startD2H), stream);
        //err = checkCuda(hipMemcpy(D2H, dev, size * sizeof(T), hipMemcpyDeviceToHost))
        err = copyCUDA(parameters, D2H, dev, hipMemcpyDeviceToHost, stream);

        hipEventRecord(*(userData.stopD2H), stream);
        nvtxRangePop();

    }

    // Synchronize CUDA events and get timers
    if (!err)
    {
        

       //hipEventSynchronize(stopH2D); //put that in a thread ??
       //hipEventSynchronize(stopD2H);
        hipStreamWaitEvent(stream, *(userData.stopH2D));
        hipStreamWaitEvent(stream, *(userData.stopD2H));
        std::cout << "X1" << std::endl;
        hipStreamAddCallback(stream, callback, &userData, 0);
        std::cout << "X2" << std::endl;

        
    }

    return err;
}