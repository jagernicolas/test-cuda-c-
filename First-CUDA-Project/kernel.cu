
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <algorithm>
#include <chrono>
#include <iostream>

class cLimitTimer
{
    std::chrono::time_point<std::chrono::steady_clock> Start;
    double Limit;
public:
    cLimitTimer(double limit) : Limit(limit)
    {
        Start = std::chrono::steady_clock::now();
    }
    ~cLimitTimer()
    {
        auto end = std::chrono::steady_clock::now();

        std::chrono::duration<double> diff = end - Start;
        if (diff.count() > Limit)
            std::cout << diff.count() << "s greater than limit " << Limit << "s\n";
        else if(diff.count() > 1.0)
            std::cout << diff.count() << "s\n";
        else
            std::cout << diff.count() * 1000 << "ms\n";

    }
};

hipError_t addWithCuda(int* B, int* A, unsigned int size);

__global__ void addKernel(int *B, int *A, unsigned int size)
{
    //1;
    for (int i = 0; i < size; i++)
         B[i] = A[i];
}

int main()
{
    auto* timer = new cLimitTimer(1000);
    const int arraySize = 50000000;
    int *A = (int*)malloc(sizeof(int) * arraySize);  
    // chrono here
    std::fill_n(A, arraySize, 1);
    int* B = (int*)malloc(sizeof(int) * arraySize);

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(B, A, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("\nresult = {%d,%d,%d,%d,%d...%d}\n",
    B[0], B[1], B[2], B[3], B[4], B[arraySize-1]);

    free(A);
    free(B);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    printf("total time: ");
    delete timer;
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *B, int *A, unsigned int size)
{
    auto* timer1 = new cLimitTimer(1000);
    int* dev_A = 0;
    int* dev_B = 0;
    hipError_t cudaStatus;
    hipEvent_t start, stop; //xxx
    hipEventCreate(&start); //xxx
    hipEventCreate(&stop); //xxx
    hipEvent_t start2, stop2;//xxx
    hipEventCreate(&start2); //xxx
    hipEventCreate(&stop2); //xxx
    hipEvent_t start3, stop3;//xxx
    hipEventCreate(&start3); //xxx
    hipEventCreate(&stop3); //xxx
    float milliseconds = 0; //XXX
    float milliseconds2 = 0; //XXX
    float milliseconds3 = 0; //XXX
    float seconds = 0; //XXX
    float seconds2 = 0; //XXX
    float seconds3 = 0; //XXX
    int* B1 = (int*)malloc(sizeof(int) * size);
    printf("initialization time: ");
    delete timer1;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;

    }

    // Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&dev_A, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_B, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    timer1 = new cLimitTimer(1000);
    hipEventRecord(start); //xxx

    cudaStatus = hipMemcpy(dev_A, A, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy dev_A, A failed!");
        goto Error;
    }
    hipEventRecord(stop); //xxx 
    printf("Host->GPU... : ");
    delete timer1;

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    timer1 = new cLimitTimer(1000);
    for (int i = 0; i < size; i++)
        B1[i] = A[i];
    printf("processing time on cpu : ");
    delete timer1;

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, 1>>>(dev_B, dev_A, size); // using thread, keep in mind to don't go over max cores.

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    hipEventRecord(start3); //xxx CUDA stream is simply a sequence of operations that are performed in order on the device
    timer1 = new cLimitTimer(1000);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    hipEventRecord(stop3); //xxx
    printf("cudaDeviceSynchronize: ");
    delete timer1;

    timer1 = new cLimitTimer(1000);
    // Copy output vector from GPU buffer to host memory.
    hipEventRecord(start2); //xxx
    cudaStatus = hipMemcpy(B, dev_B, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy B, dev_B failed!");
        goto Error;
    }

    hipEventRecord(stop2); //xxx
    printf("GPU->Host... : ");
    delete timer1;

    hipEventSynchronize(stop); //XXX wait that event stop occured (cudaEventRecord) // test if this is needed
    hipEventSynchronize(stop2); //XXX same for stop2
    hipEventSynchronize(stop3); //XXX same for stop3
    
    
    printf("\n\nCuda event timers:\n");
    hipEventElapsedTime(&milliseconds, start, stop); //XXX
    printf("Host->GPU, Ellapsed time: %fms\n", milliseconds); //XXX
    hipEventElapsedTime(&milliseconds2, start2, stop2); //XXX
    printf("GPU->Host, Ellapsed time: %fms\n", milliseconds2); //XXX
    hipEventElapsedTime(&milliseconds3, start3, stop3); //XXX
    printf("processing time on GPU, Ellapsed time: %fms\n\n", milliseconds3); //XXX // I think this measure is somehow related to what is happening on the GPU.

Error:
    hipFree(dev_A);
    hipFree(dev_B);

    return cudaStatus;
}
